#include "hip/hip_runtime.h"
/* 
 * @author: Popescu Andrei Gabriel 333CA
 * @category: CUDA GPU programming
 *
 */


#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>

#include "gpu_hashtable.hpp"

/*
 * CUDA Function for exposed HashTable API
 *
 * */

/*
 * Constant values
 *
 **/
#define LOAD_FACTOR                     0.8f
#define DEFAULT_WORKERS_BLOCK           512
#define DEFAULT_STATUS                  -1
#define FAIL                            false
#define SUCCESS                         true

/*
 *  Device functions cannot be called from host functions
 *  so basically is a API exposure problem to make this
 *  a kernel function or even a classical host function
 *
 **/

__device__ int getHash(int data, int limit) {
    return (long)abs(data) % limit;
}

__global__ void kernelInsertEntry(
        int *keys,
        int *values,
        int numKeys,
        HashTableEntry *hashTableBuckets,
        int limitSize) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > numKeys)
        return;

    int currentKey = keys[idx];
    int currentValue = values[idx];
    int hash = getHash(currentKey, limitSize);
    int status = DEFAULT_STATUS;

    /*
     * Searching from current position in hashTable to the end
     */
    for (int i = 0; i < limitSize - hash; i++) {
        status = atomicCAS(&hashTableBuckets[hash + i].HashTableEntryKey, KEY_INVALID, currentKey);

        if (status ==  DEFAULT_STATUS || status == currentKey) {
            /* Add new or replace */
            hashTableBuckets[hash + i].HashTableEntryKey = currentKey;
            hashTableBuckets[hash + i].HashTableEntryValue = currentValue;
            return;
        }
    }

}

__global__ void kernelGetEntry(
        int *keys,
        int *values,
        int numKeys,
        int limitSize,
        HashTableEntry *hashTableBuckets) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > numKeys)
        return;

    int currentKey = keys[idx];
    int hash = getHash(currentKey, limitSize);

    for (int i = 0; i < limitSize - hash; i++) {
        if (hashTableBuckets[hash + i].HashTableEntryKey == currentKey) {
            /* Insert in the values vector */
            values[idx] = hashTableBuckets[hash + i].HashTableEntryValue;
            return;
        }
    }

}

__global__ void kernelCopyHashTable(
        HashTableEntry *hashTableBucketsOrig,
        int limitSizeOrig,
        HashTableEntry *hashTableBuckets) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > limitSizeOrig)
        return;

    int status = DEFAULT_STATUS;
    int statusEmpty = -2;

    status = atomicCAS(&hashTableBuckets[idx].HashTableEntryKey, KEY_INVALID, statusEmpty);

    if (status == DEFAULT_STATUS) {
        hashTableBuckets[idx].HashTableEntryKey =
                hashTableBucketsOrig[idx].HashTableEntryKey;
        hashTableBuckets[idx].HashTableEntryValue =
                hashTableBucketsOrig[idx].HashTableEntryValue;
        return;
    }
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
    limitSize = size;
    currentSize = 0;
    cout << "[HOST] Host is allocating right now...!\n";

    hipMalloc(&hashTableBuckets, limitSize * sizeof(HashTableEntry));
    if (hashTableBuckets == 0) {
        cerr << "[HOST] Couldn't allocate memory for GpuHashTable!\n";
    }

    cout << "[HOST] Host has allocated right now...!\n";
    hipMemset(hashTableBuckets, 0, limitSize * sizeof(HashTableEntry));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
    hipFree(hashTableBuckets);
}

/* RESHAPE HASH
 */
void GpuHashTable::reshape(int numBucketsReshape) {
    HashTableEntry *hashTableBucketsReshaped;
    int newLimitSize = numBucketsReshape;

    hipMallocManaged(&hashTableBucketsReshaped, newLimitSize * sizeof(HashTableEntry));

    if (hashTableBucketsReshaped == 0) {
        cerr << "[HOST] Couldn't allocate memory for GpuHashTable Reshape!\n";
    }

    hipMemset(hashTableBucketsReshaped, 0, newLimitSize * sizeof(HashTableEntry));

    int blocks;
    if (limitSize % DEFAULT_WORKERS_BLOCK == 0)
        blocks = newLimitSize / DEFAULT_WORKERS_BLOCK;
    else
        blocks = newLimitSize / DEFAULT_WORKERS_BLOCK + 1;

    kernelCopyHashTable<<< blocks, DEFAULT_WORKERS_BLOCK >>>(hashTableBuckets, limitSize, hashTableBucketsReshaped);

    hipDeviceSynchronize();
    hipFree(hashTableBuckets);

    hashTableBuckets = hashTableBucketsReshaped;
    limitSize = newLimitSize;
}

/* INSERT BATCH
 */
bool GpuHashTable::insertBatch(int *keys, int* values, int numKeys) {
    int futureLoadFactor = (float) (currentSize + numKeys) / limitSize;
    if (futureLoadFactor > LOAD_FACTOR) {
        reshape(2 * limitSize);
    }
	
	currentSize += numKeys;
	
    int *deviceKeys;
    int *deviceValues;
    int blocks;
    if (numKeys % DEFAULT_WORKERS_BLOCK == 0)
        blocks = numKeys / DEFAULT_WORKERS_BLOCK;
    else
        blocks = numKeys / DEFAULT_WORKERS_BLOCK + 1;

    hipMallocManaged(&deviceKeys, numKeys * sizeof(int));
    hipMallocManaged(&deviceValues, numKeys * sizeof(int));

    if (deviceValues == 0 || deviceKeys == 0) {
        cerr << "[HOST] Couldn't allocate memory for device keys or values arrays!\n";
        return FAIL;
    }

    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);

    kernelInsertEntry<<< blocks, DEFAULT_WORKERS_BLOCK >>>(
            deviceKeys,
            deviceValues,
            numKeys,
            hashTableBuckets,
            limitSize
            );

    hipDeviceSynchronize();

    hipFree(deviceKeys);
    hipFree(deviceValues);

	return SUCCESS;
}

/* GET BATCH
 */
int* GpuHashTable::getBatch(int* keys, int numKeys) {
    int *deviceKeys;
    int *values;
    int *deviceValues;

    hipMallocManaged(&deviceKeys, numKeys * sizeof(int));
    hipMallocManaged(&deviceValues, numKeys * sizeof(int));
    values = (int *) malloc(numKeys * sizeof(int));

    if (deviceKeys == 0 || deviceValues == 0 || values == 0) {
        cerr << "[HOST] Couldn't allocate memory for device keys or values arrays!\n";
        return NULL;
    }

    hipMemset(deviceValues, 0, numKeys * sizeof(int));
    hipMemcpy(deviceKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);

    int blocks;
    if (numKeys % DEFAULT_WORKERS_BLOCK == 0)
        blocks = numKeys / DEFAULT_WORKERS_BLOCK;
    else
        blocks = numKeys / DEFAULT_WORKERS_BLOCK + 1;

    kernelGetEntry<<< blocks, DEFAULT_WORKERS_BLOCK >>>(
            keys,
            values,
            numKeys,
            limitSize,
            hashTableBuckets
            );

    hipDeviceSynchronize();
    hipMemcpy(values, deviceValues, numKeys * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceValues);
    hipFree(deviceKeys);
	
    return values;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
    if (currentSize != 0) {
        // No elements in HashTable
        return 0.f;
    } else {
        return (float) currentSize / limitSize;
    }
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
 