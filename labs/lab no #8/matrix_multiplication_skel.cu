

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>
#include <iostream>

#define TILE_WIDTH 16

// Task 1 - simple matrix multiplication
__global__ void matrix_multiply_simple(float *ma, float *mb, float *mc, size_t width)
{
	//TODO: calculate the row & column index of the element

	//TODO: do dot product between row of ma and column of mb

	//TODO: write result in mc
}

// Task 2 - optimized matrix multiplication
__global__ void matrix_multiply(float *ma, float *mb, float *mc, size_t width)
{
	int tx = threadIdx.x, ty = threadIdx.y;
	int bx = blockIdx.x,  by = blockIdx.y;

	//TODO: allocate 2D tiles in __shared__ memory

	//TODO: calculate the row & column index of the element

	float result = 0;

	// loop over the tiles of the input
	for(int t = 0; t < width/TILE_WIDTH; ++t) {
	
		//TODO: load tiles into __shared__ memory allocated before
		
		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue

		//TODO: do dot product between row of tile from ma and column of tile from mb

		//TODO:
		// wait until all data is loaded before allowing
		// any thread in this block to continue
	}

	//TODO: write result in mc
}

int main(void)
{
	// create a large workload so we can easily measure the
	// performance difference of both implementations

	// note that n measures the width of the matrix, not the number of total elements
	const size_t n = 1<<10;
	const dim3 block_size(TILE_WIDTH,TILE_WIDTH);
	const dim3 num_blocks(n / block_size.x, n / block_size.y);

	// generate random input on the host
	std::vector<float> host_a(n*n), host_b(n*n), host_c(n*n);
	for(int i = 0; i < n*n; ++i) {
		host_a[i] = static_cast<float>(rand()) / RAND_MAX;
		host_b[i] = static_cast<float>(rand()) / RAND_MAX;
	}

	// allocate storage for the device
	float *device_a = 0, *device_b = 0, *device_c = 0;
	hipMalloc((void**)&device_a, sizeof(float) * n * n);
	hipMalloc((void**)&device_b, sizeof(float) * n * n);
	hipMalloc((void**)&device_c, sizeof(float) * n * n);

	// copy input to the device
	hipMemcpy(device_a, &host_a[0], sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(device_b, &host_b[0], sizeof(float) * n * n, hipMemcpyHostToDevice);

	//Task 3 - measure the time spent in the kernel for simple and optimized implementation
	
	//TODO: create CUDA events for measuring kernel time
	hipEvent_t launch_begin, launch_end;

	// time many kernel launches and take the average time
	const size_t num_launches = 100;
	float average_simple_time = 0;
	std::cout << "Timing simple implementation...";
	
	for(int i = 0; i < num_launches; ++i) {
		//TODO: record CUDA event before and after the kernel launch

		matrix_multiply_simple<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);

		//TODO: Wait for launch_end event to complete

		//TODO: measure the time spent in the kernel
		float time = 0;

		average_simple_time += time;
	}
	
	average_simple_time /= num_launches;
	std::cout << " done." << std::endl;

	//now time the optimized kernel

	// time many kernel launches and take the average time
	float average_optimized_time = 0;
	std::cout << "Timing optimized implementation...";
	for(int i = 0; i < num_launches; ++i) {
		//TODO: record CUDA event before and after the kernel launch
		
		matrix_multiply<<<num_blocks,block_size>>>(device_a, device_b, device_c, n);

		//TODO: Wait for launch_end event to complete
		
		//TODO: measure the time spent in the kernel
		float time = 0;

		average_optimized_time += time;
	}
	average_optimized_time /= num_launches;
	std::cout << " done." << std::endl;

	// report the effective throughput of each kernel in GFLOPS
	// the effective throughput is measured as the number of floating point operations performed per second:
	// (one mul + one add) * N^3
	float simple_throughput = static_cast<float>(2 * n * n * n) / (average_simple_time / 1000.0f) / 1000000000.0f;
	float optimized_throughput = static_cast<float>(2 * n * n * n) / (average_optimized_time / 1000.0f) / 1000000000.0f;

	std::cout << "Matrix size: " << n << "x" << n << std::endl;
	std::cout << "Tile size: " << TILE_WIDTH << "x" << TILE_WIDTH << std::endl;

	std::cout << "Throughput of simple kernel: " << simple_throughput << " GFLOPS" << std::endl;
	std::cout << "Throughput of optimized kernel: " << optimized_throughput << " GFLOPS" << std::endl;
	std::cout << "Performance improvement: " << optimized_throughput / simple_throughput << "x" << std::endl;
	std::cout << std::endl;

	//TODO: destroy the CUDA events

	// deallocate device memory
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);

	return 0;
}

