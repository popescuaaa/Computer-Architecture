#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1: Allocate the host's arrays

    // TODO 2: Allocate the device's arrays

    // TODO 3: Check for allocation errors

    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.

    // TODO 5: Copy the host's arrays to device

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).

    // TODO 7: Copy back the results and then uncomment the checking function

    /* check_task_2(host_array_a, host_array_b, host_array_c, N); */

    // TODO 8: Free the memory
   
    return 0;
}