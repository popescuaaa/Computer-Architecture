#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

#define BUF_2M		(2 * 1024 * 1024)
#define BUF_32M		(32 * 1024 * 1024)

int main(void) {
    hipSetDevice(0);

    int *host_array_a = 0;
    int *host_array_b = 0;

    int *device_array_a = 0;
    int *device_array_b = 0;
    int *device_array_c = 0;

    // TODO 1: Allocate the host's arrays with the specified number of elements:
    // host_array_a => 32M
    // host_array_b => 32M
    host_array_a = (float *) malloc(BUF_32M);
    host_array_b = (float *) malloc(BUF_32M);

    // TODO 2: Allocate the device's arrays with the specified number of elements:
    // device_array_a => 32M
    // device_array_b => 32M
    // device_array_c => 2M

    hipMalloc( (void **) &device_array_a, BUF_32M);
    hipMalloc( (void **) &device_array_b, BUF_32M);
    hipMalloc( (void **) &device_array_c, BUF_2M);

    // Check for allocation errors
    if (host_array_a == 0 || host_array_b == 0 || 
        device_array_a == 0 || device_array_b == 0 || 
        device_array_c == 0) {
        printf("[*] Error!\n");
        return 1;
    }

    for (int i = 0; i < BUF_32M; ++i) {
        host_array_a[i] = i % 32;
        host_array_b[i] = i % 2;
    }

    printf("Before swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 3: Copy from host to device
    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    // TODO 4: Swap the buffers (BUF_2M values each iteration)
    // Hint 1: device_array_c should be used as a temporary buffer
    // Hint 2: hipMemcpy
    
    // TODO 5: Copy from device to host

    printf("\nAfter swap:\n");
    printf("a[i]\tb[i]\n");
    for (int i = 0; i < 10; ++i) {
        printf("%d\t%d\n", host_array_a[i], host_array_b[i]);
    }

    // TODO 6: Free the memory

    return 0;
}