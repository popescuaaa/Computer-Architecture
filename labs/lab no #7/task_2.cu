#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "utils/utils.h"

// TODO 6: Write the code to add the two arrays element by element and 
// store the result in another array
__global__ void add_arrays(const float *a, const float *b, float *c, int N) {
    unsigned int local_index = threadIdx.x + blockDim.x * blockIdx.x;
    if (local_index < N) {
        c[local_index] = a[local_index] + b[local_index];
    }
}

int main(void) {
    hipSetDevice(0);
    int N = 1 << 20;
    int num_bytes = N * sizeof(float);

    float *host_array_a = 0;
    float *host_array_b = 0;
    float *host_array_c = 0;

    float *device_array_a = 0;
    float *device_array_b = 0;
    float *device_array_c = 0;

    // TODO 1 & 3: Allocate the host's arrays

    host_array_a = (float *) malloc(num_bytes);
    if (host_array_a == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }
    host_array_b = (float *) malloc(num_bytes);
    if (host_array_b == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }
    host_array_c = (float *) malloc(num_bytes);
    if (host_array_c == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }

    // TODO 2 & 3: Allocate the device's arrays
    hipMalloc( (void **) &device_array_a, num_bytes);
    if (device_array_a == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }
    hipMalloc( (void **) &device_array_b, num_bytes);
    if (device_array_b == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }
    hipMalloc( (void **) &device_array_c, num_bytes);
    if (device_array_c == 0) {
        printf("BAD_ALLOC");
        exit(12);
    }
    
    // TODO 4: Fill array with values; use fill_array_float to fill
    // host_array_a and fill_array_random to fill host_array_b. Each
    // function has the signature (float *a, int n), where n = number of elements.
    fill_array_float(host_array_a, N);
    fill_array_random(host_array_b, N);

    // TODO 5: Copy the host's arrays to device
    hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

    // TODO 6: Execute the kernel, calculating first the grid size
    // and the amount of threads in each block from the grid
    // Hint: For this execise the block_size can have any value lower than the
    //      API's maximum value (it's recommended to be close to the maximum
    //      value).

    const size_t block_size = 256;
    size_t blocks_no = num_elements / block_size;

    if (num_elements % block_size) 
        ++blocks_no;
    
    add_arrays<<<blocks_no, block_size>>>(device_array_a, device_array_b, device_array_c, num_elements);

    // TODO 7: Copy back the results and then uncomment the checking function
    hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

    check_task_2(host_array_a, host_array_b, host_array_c, N);

    // TODO 8: Free the memory

    free(host_array_a);
    free(host_array_b);
    free(host_array_c);
    hipFree(device_array_a);
    hipFree(device_array_b);
    hipFree(device_array_c);

    return 0;
}