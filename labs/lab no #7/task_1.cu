#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

// ~TODO 3~
// Modify the kernel below such as each element of the 
// array will be now equal to 0 if it is an even number
// or 1, if it is an odd number
__global__ void kernel_parity_id(int *a, int N) {
	
}

// ~TODO 4~
// Modify the kernel below such as each element will
// be equal to the BLOCK ID this computation takes
// place.
__global__ void kernel_block_id(int *a, int N) {

}

// ~TODO 5~
// Modify the kernel below such as each element will
// be equal to the THREAD ID this computation takes
// place.
__global__ void kernel_thread_id(int *a, int N) {

}

int main(void) {
    int nDevices;

    // Get the number of CUDA-capable GPU(s)
    hipGetDeviceCount(&nDevices);

    // ~TODO 1~
    // For each device, show some details in the format below, 
    // then set as active device the first one (assuming there
    // is at least CUDA-capable device). Pay attention to the
    // type of the fields in the hipDeviceProp_t structure.
    //
    // Device number: <i>
    //      Device name: <name>
    //      Total memory: <mem>
    //      Memory Clock Rate (KHz): <mcr>
    //      Memory Bus Width (bits): <mbw>
    // 
    // Hint: look for hipGetDeviceProperties and hipSetDevice in
    // the Cuda Toolkit Documentation. 
    hipDeviceProp_t properties;

    for (int i = 0; i < nDevices; ++i) {
	hipGetDeviceProperties(&properties, i);
	
	printf("\t Device properties: << %d >> \n\n", i);
	printf("\t\t\n Device name: %s \n", properties.name);
	printf("\t\t\n Memory Clock Rate: %d \n", properties.clockRate);
	printf("\t\t\n Memory Bus Width: %d \n", properties.memoryBusWidth);

    }

    // ~TODO 2~
    // With information from example_2.cu, allocate an array with
    // integers (where a[i] = i). Then, modify the three kernels
    // above and execute them using 4 blocks, each with 4 threads.
    // Hint: num_elements = block_size * block_no (see example_2)
    //
    // You can use the fill_array_int(int *a, int n) function (from utils)
    // to fill your array as many times you want.
  
    // ~TODO 3~
    // Execute kernel_parity_id kernel and then copy from 
    // the device to the host; call hipDeviceSynchronize()
    // after a kernel execution for safety purposes.
    //
    // Uncomment the line below to check your results

    /* check_task_1(3, host_array); */

    // ~TODO 4~
    // Execute kernel_block_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results

    /* check_task_1(4, host_array); */

    // ~TODO 5~
    // Execute kernel_thread_id kernel and then copy from 
    // the device to the host;
    //
    // Uncomment the line below to check your results

    /* check_task_1(5, host_array); */

    // TODO 6: Free the memory
    
    return 0;
}
