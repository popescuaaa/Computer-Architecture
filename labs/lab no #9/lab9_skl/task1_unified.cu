#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

#define NUM_ELEM    (16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void) {
    float *host_a = 0;
    float *host_b = 0;

    // TODO-unified: Alocare memorie unificata

    fill_array_random(host_b, NUM_ELEM);

    kernel_compute<<<NUM_ELEM / 256, 256>>> (host_a, host_b, NUM_ELEM);
    
    // TODO-unified: Dealocare memorie unificata
    
    return 0;
}

int compute_NoUnifiedMem(void) {
    float *device_a = 0;
    float *device_b = 0;
    float *host_a = 0;
    float *host_b = 0;

    host_a = (float *) malloc(NUM_ELEM * sizeof(float));
    host_b = (float *) malloc(NUM_ELEM * sizeof(float));
    
    // TODO-plain: Alocare memorie (GPU/VRAM)
    if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
        printf("[HOST] Couldn't allocate memory\n");
    	return 1;
    }

    fill_array_random(host_b, NUM_ELEM);
    
    // TODO-plain: Copiere date host_b (CPU/RAM) => device_b (GPU/VRAM)

    kernel_compute<<<NUM_ELEM / 256, 256>>> (device_a, device_b, NUM_ELEM);
    
    // TODO-plain: Copiere device_b (GPU/VRAM) => date host_b (CPU/RAM)
    
    free(host_a);
    free(host_b);

    // TODO-plain: Dealocare memorie (GPU/VRAM)
    
    return 0;
}

int main(void) {
#ifdef NO_UNIFIED_MEMORY
    compute_NoUnifiedMem();
#else
    compute_UnifiedMem();
#endif
}
