#include "hip/hip_runtime.h"
#include <stdio.h>
#include "utils/utils.h"

#define NUM_ELEM	(16 * 1024 * 1024)

__global__ void kernel_compute(float* a, float* b, int N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	a[idx] = sin(b[idx]) * (1 - cos(b[idx]));
}

int compute_UnifiedMem(void)
{
	float *host_a = 0;
	float *host_b = 0;

	// TODO-unified: Alocare memorie unificata
	hipMallocManaged((void **)&host_a, NUM_ELEM * sizeof(*host_a));
	hipMallocManaged((void **)&host_b, NUM_ELEM * sizeof(*host_b));

	if (!host_a || !host_b) {
		fprintf(stderr, "[HOST & DEVICE] hipMallocManaged failed\n");
		return 1;
	}

	fill_array_random(host_b, NUM_ELEM);

	kernel_compute<<<NUM_ELEM / 256, 256>>> (host_a, host_b, NUM_ELEM);

	// TODO-unified: Dealocare memorie unificata
	hipFree(host_a);
	hipFree(host_b);
	
	return 0;
}

int compute_NoUnifiedMem(void)
{
	float *device_a = 0;
	float *device_b = 0;
	float *host_a = 0;
	float *host_b = 0;

	host_a = (float *) malloc(NUM_ELEM * sizeof(float));
	host_b = (float *) malloc(NUM_ELEM * sizeof(float));
	
	// TODO-plain: Alocare memorie (GPU/VRAM)
	hipMalloc((void **)&device_a, NUM_ELEM * sizeof(*device_a));
	hipMalloc((void **)&device_b, NUM_ELEM * sizeof(*device_b));
	
	if (host_a == 0 || host_b == 0 || device_a == 0 || device_b == 0) {
		printf("[HOST] Couldn't allocate memory\n");
		return 1;
	}

	fill_array_random(host_b, NUM_ELEM);
	
	// TODO-plain: Copiere date host_b (CPU/RAM) => device_b (GPU/VRAM)
	hipMemcpy(device_b, host_b, NUM_ELEM, hipMemcpyHostToDevice);

	kernel_compute<<<NUM_ELEM / 256, 256>>> (device_a, device_b, NUM_ELEM);

	// TODO-plain: Copiere device_b (GPU/VRAM) => date host_b (CPU/RAM)
	hipMemcpy(host_a, device_a, NUM_ELEM, hipMemcpyDeviceToHost);

	free(host_a);
	free(host_b);

	// TODO-plain: Dealocare memorie (GPU/VRAM)
	hipFree(device_a);
	hipFree(device_b);
	
	return 0;
}

int main(void)
{
#ifdef NO_UNIFIED_MEMORY
	return compute_NoUnifiedMem();
#else
	return compute_UnifiedMem();
#endif
}